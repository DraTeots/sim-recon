#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"


__global__ void
GPUUniform_kernel(GPU_AMP_PROTO)
{

  WCUComplex ans = { 1, 0};  
  pcDevAmp[GPU_THIS_EVENT] = ans;

}

void
GPUUniform_exec(dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO)
{
  GPUUniform_kernel<<< dimGrid, dimBlock >>>(GPU_AMP_ARGS);
}

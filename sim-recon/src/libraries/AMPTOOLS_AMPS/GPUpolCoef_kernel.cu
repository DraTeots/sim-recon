#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"





__global__ void
GPUpolCoef_kernel(GPU_AMP_PROTO , int polBeam, GDouble polFrac)
{
  int pol=(polBeam==1 ? +1 : -1); // y and x-pol. respectively

  //(1+g) for x-pol, (1-g) for y-pol
  WCUComplex ans = { sqrt((1.0-pol*polFrac)*0.5), 0 };
  
  pcDevAmp[GPU_THIS_EVENT] = ans;

}

void
GPUpolCoef_exec(dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
                   int polBeam, GDouble polFrac)
{
  GPUpolCoef_kernel<<< dimGrid, dimBlock >>>
    ( GPU_AMP_ARGS, polBeam, polFrac );

}

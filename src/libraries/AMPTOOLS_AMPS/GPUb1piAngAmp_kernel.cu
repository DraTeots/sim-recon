#include "hip/hip_runtime.h"
/*
 *  GPUb1piAngAmp_kernel.cu
 *
 */


#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "GPUUtils/lorentzBoost.cuh"
#include "GPUUtils/threeVector.cuh"
#include "GPUUtils/wignerD.cuh"
#include "GPUUtils/clebsch.cuh"

#include "AMPTOOLS_AMPS/breakupMomentum.cuh"
#include "AMPTOOLS_AMPS/barrierFactor.cuh"


#define ADD4(a,b) { a[0]+b[0], a[1]+b[1], a[2]+b[2], a[3]+b[3] }

#define MASS(v)   (G_SQRT(v[0]*v[0]-v[1]*v[1]-v[2]*v[2]-v[3]*v[3]))

#define Nterm(J)  (G_SQRT((2*J+1)/(4*M_PI)))


// Macro to ease definition of loops
#define LOOP(INDEX,START,END,INC) for (int INDEX=START;INDEX<=END;INDEX+=INC)


static __device__ void //note: 4-vector input presumed
rotateZ( GDouble* v, GDouble phi ){
  GDouble sinphi = G_SIN(phi);
  GDouble cosphi = G_COS(phi);
  GDouble tx;
  tx   = v[1] * cosphi - v[2] * sinphi;
  v[2] = v[2] * cosphi + v[1] * sinphi;
  v[1] = tx;
}

static __device__ void //note: 4-vector input presumed
rotateY ( GDouble* v, GDouble theta) {
  double sinphi = G_SIN(theta);
  double cosphi = G_COS(theta);
  double tz;
  tz = v[3] * cosphi - v[1] * sinphi;
  v[1] = v[1] * cosphi + v[3] * sinphi;
  v[3] = tz;
}

static __device__ GDouble  //note: 3-vector input presumed
theta( GDouble* pv ){
  GDouble r= G_SQRT(pv[0]*pv[0] + pv[1]*pv[1]);
  return ( ( pv[2] == 0 ) && ( r == 0 ) ? 0 : G_ATAN2( r , pv[2] ) );
}


static __device__ void
MoveToRF(GDouble *parent, GDouble *daughter)
{
  GDouble *par3vec=parent+1;
  rotateZ( daughter , -phi(par3vec) );
  rotateY( daughter , -theta(par3vec) );

  GDouble beta[]={0,0, -G_SQRT(dot(par3vec,par3vec))/parent[0]};
  boost( daughter , beta );

}



static __device__ WCUComplex
BreitWigner_loc(GDouble m0, GDouble Gamma0, int L,
                GDouble *P1, GDouble *P2)
{
  
  GDouble  Ptot[4] = ADD4(P1, P2);  
  GDouble m  = MASS(Ptot);
  GDouble mass1 = MASS(P1);
  GDouble mass2 = MASS(P2);
  
  
  // assert positive breakup momenta     
  GDouble q0 = fabs( breakupMomentum(m0, mass1, mass2) );
  GDouble q  = fabs( breakupMomentum(m,  mass1, mass2) );
  
  GDouble F0 = L==0 ? 1.0 : barrierFactor(q0, L);
  GDouble F  = L==0 ? 1.0 : barrierFactor(q,  L);
  
  GDouble width_coef=Gamma0*(m0/m);
  //GDouble qq0=q/q0;
  //GDouble width_qdep = (L==0 ? qq0 : (L==1 ? qq0*qq0*qq0 : pow(qq0,2*L+1)))*((F*F)/(F0*F0));
  GDouble width_qdep = q/q0  * (F*F)/(F0*F0);
  //GDouble num_qdep = (L==0 ? q : (L==1 ? q*q*q : pow(q,2*L+1)))*(F*F);
  GDouble num_qdep = q*(F*F);
  
  GDouble width = width_coef * width_qdep;
  
  //complex<GDouble> bwtop(m0 * width, 0.0 );
  WCUComplex bwtop = { G_SQRT(m0*width_coef) * num_qdep, 0 };
  
  WCUComplex bwbottom =  { m0*m0 - m*m  , -1.0 * ( m0 * width ) };
  
  return  ( bwtop / bwbottom );
  
}



// 2012-07-12 JR  Remove polFrac in parameter list.
__global__ void
GPUb1piAngAmp_kernel( GPU_AMP_PROTO , int polBeam, 
  int J_X, int Par_X, int L_X, int I_X, int epsilon_R, int Iz_b1, int Iz_pi,
  GDouble u_rho_1, GDouble u_rho_3, GDouble u_omega_1, GDouble u_omega_3,
  GDouble u_b1_0, GDouble u_b1_2, 
  GDouble G0_omega, GDouble G0_b1, bool orthocheck) 
{
  // Calculate event for this thread.
  int iEvent = GPU_THIS_EVENT;

  WCUComplex CZero = { 0, 0 };
  WCUComplex i =     { 0, 1 };
  WCUComplex COne =  { 1, 0 };

  int pol=(polBeam==1 ? +1 : -1); // y and x-pol. respectively
  
  if (J_X==0 && Par_X*pol*epsilon_R==-1) {
    pcDevAmp[iEvent] = CZero;
    return;
  }



  int m_X;
  GDouble u_rho, u_omega, u_b1;
  GDouble InvSqrt2 = 1.0/G_SQRT(2.0);
  GDouble m0_rho   = 0.775;
  GDouble G0_rho   = 0.149;
  GDouble m0_omega = 0.783;
  GDouble m0_b1    = 1.223;
  bool useCutoff   = true;
  bool isZero;



  //  Copy four-vectors for this thread from global memory.
  //  2012-05-19 JR  rhos_pip0,omega0,rho0 added for use
  //                 in BreitWigner_loc() below.
  GDouble  beam     [4] = GPU_P4(0);
  GDouble  recoil   [4] = GPU_P4(1);
  GDouble  Xs_pi    [4] = GPU_P4(2);
  GDouble  b1s_pi   [4] = GPU_P4(3);
  GDouble  omegas_pi[4] = GPU_P4(4);
  GDouble  rhos_pim [4] = GPU_P4(5);
  GDouble  rhos_pip [4] = GPU_P4(6);
  GDouble  rhos_pip0[4] = GPU_P4(6);

  //  Make four-vector sums
  GDouble  rho   [4] = ADD4(rhos_pip, rhos_pim );
  GDouble  rho0  [4] = ADD4(rhos_pip, rhos_pim );
  GDouble  omega [4] = ADD4(rho,     omegas_pi);
  GDouble  omega0[4] = ADD4(rho,     omegas_pi);
  GDouble  b1    [4] = ADD4(omega,   b1s_pi);

  //  Store mass of b1; for other vectors we can calculate mass on the fly.
  GDouble b1mass = MASS(b1);

  //  Is this term zero?
  if (useCutoff) {
      isZero  = MASS(rho)+0.135            > m0_omega+3*G0_omega;
      isZero |= fabs(MASS(omega)-m0_omega) > 3*G0_omega;
      isZero |= fabs(b1mass-m0_b1)         > 3*G0_b1;
      isZero |= b1mass                     < (m0_omega - 3*G0_omega);
      // Zero amplitude
      if (isZero) {
         pcDevAmp[iEvent] = CZero;
         return;
      }
  }

  // Continue to Calculate amplitude
  GDouble  X[4] = ADD4(b1,      Xs_pi);

  GDouble q = breakupMomentum( MASS(X), b1mass, MASS(Xs_pi) );

  GDouble alpha = phi( &(recoil[1]) );

  //  NOTE: Values of beam and recoil are changed below.
  boostToRest (beam,   X);
  boostToRest (recoil, X);

  //  Define new coordinate system with 
  //    - beam parallel to z direction
  //    - recoil in the x,z plain (i.e., y is normal to recoil and beam)
  //    - y is normal to beam and recoil.
  GDouble zGJ[3] = { beam[1], beam[2], beam[3] };
  makeUnit( zGJ );
  
  GDouble yGJ[3] = { -recoil[1], -recoil[2], -recoil[3] };
  cross( yGJ, zGJ );
  makeUnit( yGJ );
  
  GDouble xGJ[3] = { yGJ[0], yGJ[1], yGJ[2] };
  cross( xGJ, zGJ );

  //particles to rest frames of their parents
  boostToRest (b1,      X);
  boostToRest (omega,   X);
  boostToRest (rho,     X);
  boostToRest (rhos_pip, X);
 
  // Note that in this form of the cascade of boosts, we are not
  // saving the 4-vecs in their intermediate RF, but going sequentially
  // straight to their immediate parent's RF.
  MoveToRF(b1,omega);
  MoveToRF(b1,rho);      MoveToRF(omega,rho);
  MoveToRF(b1,rhos_pip); MoveToRF(omega,rhos_pip); MoveToRF(rho,rhos_pip);
  
  GDouble *b1_3vec=b1+1;
  GDouble ang_b1[]={dot(b1_3vec, xGJ),
                    dot(b1_3vec, yGJ),
                    dot(b1_3vec, zGJ)};
  GDouble b1_XRF_cosTheta = cosTheta(ang_b1);
  GDouble b1_XRF_phi      = phi(ang_b1);
   
  GDouble rho_omegaRF_cosTheta = cosTheta(rho+1);
  GDouble rho_omegaRF_phi      = phi(rho+1);
  GDouble rhos_pip_rhoRF_cosTheta = cosTheta(rhos_pip+1);
  GDouble rhos_pip_rhoRF_phi      = phi(rhos_pip+1);
  GDouble omega_b1RF_cosTheta     = cosTheta(omega+1);
  GDouble omega_b1RF_phi          = phi(omega+1);


  // SUMMATION GUIDE:
  // notation meant to resemble TeX symbols in derivation
  // exception: pol = \epsilon_\gamma
  // l -> lambda, indicating helicity
  // u_[particle](q.n.) -> amplitude strength coefficient 

  int l_R_lim     = J_X + 1;
  
  //shortcut:  CB(L_X, J_b1, 0, l_b1 ; J_X, l_b1) vanishes when
  //  = CB(1, 1, 0, 0 ; 1, 0),  so omit l_b1=0 when J_X=L_X=1
  int l_b1_inc    = L_X==1 && J_X==1 ? 2 : 1;
  
  // restrict omega decay to just p wave
  int L_omega_lim = 1; // set to 3 to allow F wave
  int L_Rsign_lim;
  
  GDouble cosAlpha=G_COS(alpha), sinAlpha=G_SIN(alpha);
  WCUComplex expFact = {cosAlpha, sinAlpha};
  WCUComplex expFact_conj = {cosAlpha, -sinAlpha};
  
  WCUComplex ThelSum = { 0 , 0 };

  //  Setup dependent loop limits
  LOOP(l_gamma, -1, 1, 2) {

    
    LOOP(l_R, 0, l_R_lim, 1) {
      if(l_R==0 && epsilon_R==-1) continue;

      //summing positive and negative helicity terms of R's reflectivity state
      L_Rsign_lim = l_R > 0 ? -1 : +1;


      LOOP(l_Rsign, L_Rsign_lim, 1, 2) {

        m_X = l_gamma - l_Rsign * l_R;
        if (m_X==0) {
          //testing for cancelation in |J 0>+pol*P*epsilon_R*(-1)^J|J 0>
          if(Par_X*pol*epsilon_R == (J_X % 2 ==0 ? -1:+1)) continue;
        } else {
          //enforcing that the selected projection <= vector magnitude 
          if( abs(m_X)>J_X) continue; 
        }
        
        
        WCUComplex l_b1DepTerm = {0,0};
        LOOP(l_b1, -1,1,l_b1_inc) {
          

          WCUComplex L_b1DepTerm = {0,0};

          LOOP(L_b1,0,2,2) {
            
          
            WCUComplex l_omegaDepTerm = {0,0};

            LOOP(l_omega,-1,1,1) {
              
              WCUComplex L_omegaDepTerm = {0,0};
              LOOP(L_omega, 1, L_omega_lim, 2) { 
                
                WCUComplex J_rhoDepTerm = {0,0};
                LOOP(J_rho, 1, L_omega_lim, 2) {
                  
                  //enforces triang. ineq. betw. J_omega=1, J_rho and L_omega
                  // in effect, L_omega and J_rho take identical values
                  if( abs(J_rho-L_omega) > 1) continue; 
                  
                  
                  WCUComplex l_rhoDepTerm = {0,0};
                  LOOP(l_rho,-1,1,1) {
                    //shortcut CB(1,1,0,0;1,0)=0
                    if(L_omega==1 && J_rho==1 && l_rho==0) continue;
                    
                    l_rhoDepTerm += 
                      Conjugate(wignerD(1, l_omega, l_rho, 
                                        rho_omegaRF_cosTheta, rho_omegaRF_phi))
                      * clebsch(L_omega, 0, J_rho, l_rho, 1, l_rho)
                      * Y(J_rho, l_rho, rhos_pip_rhoRF_cosTheta, rhos_pip_rhoRF_phi);
                  }
                  
                  u_rho = J_rho==1 ? u_rho_1 : (J_rho==3 ? u_rho_3 : 0);
                  J_rhoDepTerm += u_rho * l_rhoDepTerm * 
                    BreitWigner_loc(m0_rho,G0_rho, J_rho,rhos_pip0,rhos_pim);
                }
                
                J_rhoDepTerm *= BreitWigner_loc(m0_omega, G0_omega, L_omega, omegas_pi,rho0);
                
                u_omega = L_omega==1 ? u_omega_1 : (L_omega==3 ? u_omega_3 : 0);
                L_omegaDepTerm += u_omega * J_rhoDepTerm * Nterm(L_omega);
              }
              
              l_omegaDepTerm += L_omegaDepTerm * 
                clebsch(L_b1, 0, 1, l_omega, 1, l_omega) *
                Conjugate(wignerD(1, l_b1, l_omega, 
                                  omega_b1RF_cosTheta, omega_b1RF_phi));
            }
            
            l_omegaDepTerm *= BreitWigner_loc(m0_b1, G0_b1, L_b1, b1s_pi, omega0);
            
            u_b1 = L_b1==0 ? u_b1_0 : (L_b1==2 ? u_b1_2 : 0); 
            L_b1DepTerm += u_b1 * l_omegaDepTerm * Nterm(L_b1);
          }
          
          l_b1DepTerm += L_b1DepTerm *
            Conjugate(wignerD(J_X, m_X, l_b1, b1_XRF_cosTheta, b1_XRF_phi)) *
            clebsch(L_X, 0, 1, l_b1, J_X, l_b1);
        }
        
        ThelSum += l_b1DepTerm  
          //to account for |eps_g> ~ (|1,-1>exp(-ia)-pol|1,+1>exp(ia)) 
          * (l_gamma==1 ? (-pol)*expFact : expFact_conj)
          //Assemble reflectivity eigenvector with epsilon_X=pol*epslion_R
          * (GDouble) (m_X<0 ? Par_X*pol*epsilon_R*((J_X-m_X) % 2 == 0 ? +1:-1) : 1) 
          * (GDouble) (m_X == 0 ? 1.0 : InvSqrt2 )
          // to apply th(l_R) reflectivity state prefactor: 
          // m=0: 1/2  m>0: 1/sqrt(2)  m<0: 0 (last just skipped in this sum)  
          * (GDouble) (l_R > 0 ? InvSqrt2 : 1.0 )
          //apply coefficients to the reflectivity basis terms:
          * (GDouble) (l_Rsign==1 ? 1 : epsilon_R)
          ; //v(*epsilon_R) *     
        
      }
    }
  }
  

  ThelSum *= Nterm(L_X) * 
    // barrier factor
    (GDouble)(L_X==0 ? 1.0 : (L_X==1 ? q : G_POW(q,L_X))) *
    // to apply polarization fraction weights: OBSOLETE! moved to polCoef class
    // (GDouble)G_SQRT((1.0-pol*polFrac)*0.5) * //(1+g) for x-pol, (1-g) for y-pol   
    (pol==1 ? i : COne)*InvSqrt2 * //to account for |eps_g> ~ sqrt(-eps/2)
    clebsch(1, Iz_b1, 1, Iz_pi, I_X, Iz_b1 + Iz_pi);
  pcDevAmp[iEvent] = ThelSum;
  
}


void
GPUb1piAngAmp_exec(dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
                   int polBeam, //GDouble polFrac,
                   int J_X, int Par_X, int L_X, int I_X, int epsilon_R, 
                   int Iz_b1, int Iz_pi,
                   GDouble u_rho_1, GDouble u_rho_3, 
                   GDouble u_omega_1, GDouble u_omega_3,
                   GDouble u_b1_0, GDouble u_b1_2, 
                   GDouble G0_omega, GDouble G0_b1, bool orthocheck)
{  
  GPUb1piAngAmp_kernel<<< dimGrid, dimBlock >>>
    ( GPU_AMP_ARGS, polBeam,
      J_X, Par_X, L_X, I_X, epsilon_R, Iz_b1, Iz_pi,
      u_rho_1, u_rho_3, u_omega_1, u_omega_3, u_b1_0, u_b1_2, 
      G0_omega, G0_b1, orthocheck ); 

}

